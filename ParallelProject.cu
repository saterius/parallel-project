#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <string>
#include <sstream>
#include <vector>
#include <algorithm>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
using namespace std;
using namespace thrust;

// #define DocNum 3
// #define Classes 2
// #define DocWords 3
// #define DocClass_0 1
// #define DocClass_1 2

#define DocNum 10
#define Classes 2
#define DocWords 20
#define DocClass_0 5
#define DocClass_1 5

__host__ int isin(host_vector<string> vocab, string f) // just check that is string f in vector vocab?
{

    // cout << "debug " << f << endl;
    if (!vocab.empty())
    {
        // cout << "vocab not empty" << endl;
        for (int i = 0; i < vocab.size(); i++)
        {

            if (vocab[i].compare(f) == 0)
            {

                return i;
            }
        }
    }

    return -1;
};

__host__ int isinclassify(host_vector<string> &vocab, string Word) {
    for (int i = 0; i < vocab.size(); i++) {
        if (vocab[i].compare(Word) == 0) {
            return i;
        }
    }
    return -1;
}

__host__ void translateDoc( host_vector<string> vocabList,host_vector<string> docs, int* docWord_arr) {
    
    int index = 0;
    for (int i = 0; i < docs.size(); i++) {
        stringstream ssin(docs[i]);

        string word;
        while (ssin >> word)
        {
            docWord_arr[index] = isin(vocabList, word);
            index++;
        }
    }
}

__host__ void getVocab(host_vector<string> &docList, host_vector<string> &vocabList) {

    for (int i = 0; i < docList.size(); i++) {
        stringstream ssin(docList[i]);

        string word;
        // printf("%s\n", word);
        while (ssin >> word) {
            if (isin(vocabList, word) == -1){
                vocabList.push_back(word);
            }
        }
    }
    // for (int i = 0; i < DocNum; i++)
    // {

    //     stringstream ssin(docList[i]);

    //     string word;
    //     printf("%s\n", word);
    //     while (ssin >> word)
    //     {

    //         if (isin(vocabList, word) == -1)
    //         {

    //             vocabList.push_back(word);
    //         }
    //     }
    // }
}

__global__ void term_ClassN(int * doc, int * termInClass, int nDoc) {
    int tid = threadIdx.x;

    // printf("this is from term_ClassN thread %d\n", tid);

    for (int j = 0; j < nDoc*DocWords; j++) {
        
        if (tid == doc[j]) {
            // printf("thread id %d and doc word is %d\n",tid, doc[j]);
            termInClass[tid] = termInClass[tid] + 1;
        }
    }

}

__global__ void find_posterior(int * termInClass, int * nDoc_class, double * posteriorProb) {
    int tid = threadIdx.x;

    double pos = ((termInClass[tid] + 1) * 1.0) / ((*nDoc_class + 2) * 1.0);
    // printf("this is thread %d and pos is %lf add arr index %d\n",tid,pos,tid * (*cur_class));
    posteriorProb[tid] = pos;
    
}

__host__ void translateDocClassify(host_vector<string> &vocabList, string doc, int* docWord_arr) {

    string eachword;
    string Word[DocWords];
  
    // making a string stream
    stringstream ssin(doc);
  
    // Read and print each word.
    int i = 0;
    while (ssin >> eachword) {
        // cout << eachword << endl;
        // Word.push_back(eachword);
        docWord_arr[i] = isinclassify(vocabList, eachword);
        // cout << eachword << endl;
        Word[i] = eachword;
        i++;
    }
}

__global__ void classifyperthread(int *d_in, int *d_out, int* docWord_arr, int sizeofVocab, double *d_posteriorProb_class0,
                                    double *d_posteriorProb_class1) {

    int id = threadIdx.x;

    bool donthave = true;
    for (int i = 0; i < Classes; i++) {
        for (int j = 0; j < DocWords; j++) {
            if (d_in[j] == id) {
                if (i == 0) {
                    d_posteriorProb_class0[id] = d_posteriorProb_class0[id] * d_posteriorProb_class0[id]; 
                    // printf("class %d posteriorProb = %f\n", i, d_posteriorProb_class0[id]);
                    donthave = false;
                    break;
                }
                else {
                    d_posteriorProb_class1[id] = d_posteriorProb_class0[id] * d_posteriorProb_class0[id]; 
                    // printf("class %d posteriorProb = %f\n", i, d_posteriorProb_class1[id]);
                    donthave = false;
                    break;
                }
            }
        }
        if (donthave) {
            if (i == 0) {
                d_posteriorProb_class0[id] = d_posteriorProb_class0[id] * (1-(d_posteriorProb_class0[id]));
                // printf("class %d posteriorProb = %f\n", i, d_posteriorProb_class0[id]);
            }
            else {
                d_posteriorProb_class1[id] = d_posteriorProb_class0[id] * (1-(d_posteriorProb_class0[id]));
                // printf("class %d posteriorProb = %f\n", i, d_posteriorProb_class1[id]);
            }
        }
    }
}

__host__ int findMax(double *priorProb, double *posteriorProb_class0, double *posteriorProb_class1, int sizeofVocab) {
    vector<double> prob;
    for (int i = 0; i < Classes; i++) {
        for (int j = 0; j <= sizeofVocab; j++) {
            if (j == 0) {
                if (i == 0) {
                    prob.push_back(posteriorProb_class0[j]);
                    // prob.push_back(priorProb[i] * posteriorProb_class0[j]);
                }
                else {
                    prob.push_back(posteriorProb_class1[j]);
                    // prob.push_back(priorProb[i] * posteriorProb_class1[j]);
                }
            }
            else if (j < sizeofVocab) {
                if (i == 0) {
                    prob[i] *= posteriorProb_class0[j];
                }
                else {
                    prob[i] *= posteriorProb_class1[j];
                }
            }
            else if (j == sizeofVocab) {
                if (i == 0) {
                    prob[i] = prob[i] * priorProb[i];
                }
                else {
                    prob[i] = prob[i] * priorProb[i];
                }
            }
        }
    }
    cout << prob[0] << " " << prob[1] << endl;
    if (prob[0] > prob[1])
        return 0;
    else
        return 1;
}

int main() {

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    host_vector<string> c_0;
    host_vector<string> c_1;

    //Test dataset
    // c_0.push_back("animal dog bird");
    // c_1.push_back("item box ball");
    // c_1.push_back("bottle item bomb");
    // string doc = "item bottle bird";
    // string doc = "animal dog bird";

    c_0.push_back("eligator hosting server we have hosting that can serve you Just paid 20 dollars per month for hosting your web");
    c_0.push_back("explore our selection of local favorites with 0 dollars delivery fee for your first month 10 dollars order minimum terms");
    c_0.push_back("need graphic design help in just a few clicks you can scale your creative output by hiring our pro designer");
    c_0.push_back("so your business is up and running now what grow with a marketing crm that gets smarter as you go");
    c_0.push_back("start and grow your business with shopify turn what you love into what you sell try shopify for free today");
    // c_0.push_back("looking for new glasses answer a few quick questions and we will suggest some great looking frames for you free");


    c_1.push_back("today I feel like I want to sleep all day I just wanna lay in my bed and go sleep");
    c_1.push_back("this week is rainy everyday I have to take my umbrella everyday it make me annoy sometimes when I walk");
    c_1.push_back("I am so tired I just want to rest in my vacation time go see outside not sit in table");
    c_1.push_back("she go to market to buy some pills but when she went out she forgot her wallet at her home");
    c_1.push_back("I am so tired now so I want to go to bed because I feel like I am not ok");

    // string doc = "create your website for your business just 399 dollars per month you can create your beautiful website for your business";
    // string doc = "I think I will go to sleep so do not disturb me I so tired now leave me alone please";
    // string doc = " I am so tired now so I want to go to bed because I feel like I am not ok";
    // string doc = "I feel like I am so tired I want to sleep everyday when I sleep in my bed feel good";
    string doc = "I am so tired now so I want to go to bed because I feel like I am not ok";
    // ***class 0 is ads class 1 is not ads***

    host_vector<string> vocabList;
    double priorProb[Classes];

    priorProb[0] = ((DocClass_0 + 1) * 1.0) / (((DocClass_0 + DocClass_1) + 2) * 1.0);
    priorProb[1] = ((DocClass_1 + 1) * 1.0) / (((DocClass_0 + DocClass_1) + 2) * 1.0);

    getVocab(c_0, vocabList);
    getVocab(c_1, vocabList);

    int class_0_arr[DocClass_0*DocWords];
    int class_1_arr[DocClass_1*DocWords];

    int termInClass_0[DocNum*DocWords];
    int termInClass_1[DocNum*DocWords];

    for (int t = 0; t < DocNum*DocWords; t++) {    // set value in termInClass to 0 for count in function
        termInClass_0[t] = 0;
        termInClass_1[t] = 0;
    }

    translateDoc(vocabList, c_0, class_0_arr);
    translateDoc(vocabList, c_1, class_1_arr);

    // kernel ---------------------------------------------------
    int * d_doc_array, *d_termInClass_0,*d_termInClass_1 ;


    // class 0

    hipMalloc((void **) &d_doc_array, DocClass_0*DocWords*sizeof(int));
    hipMalloc((void **) &d_termInClass_0, DocNum*DocWords*sizeof(int));
            
    hipMemcpy(d_doc_array, &class_0_arr, DocClass_0*DocWords*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_termInClass_0, &termInClass_0, DocNum*DocWords*sizeof(int), hipMemcpyHostToDevice);

	hipEventRecord(start);

    term_ClassN<<<1,DocNum*DocWords>>>(d_doc_array, d_termInClass_0,DocClass_0);

    hipMemcpy(&termInClass_0, d_termInClass_0, DocNum*DocWords*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_doc_array);
    // hipFree(d_termInClass_0);

    // ---------------

    // class 1

    hipMalloc((void **) &d_doc_array, DocClass_1*DocWords*sizeof(int));
    hipMalloc((void **) &d_termInClass_1, DocNum*DocWords*sizeof(int));
            
    hipMemcpy(d_doc_array, &class_1_arr, DocClass_1*DocWords*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_termInClass_1, &termInClass_1, DocNum*DocWords*sizeof(int), hipMemcpyHostToDevice);

    term_ClassN<<<1,DocNum*DocWords>>>(d_doc_array, d_termInClass_1,DocClass_1);

    hipMemcpy(&termInClass_1, d_termInClass_1, DocNum*DocWords*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_doc_array);
    // hipFree(d_termInClass_1);


    int * d_nDoc_class ;
    
    double * d_posteriorProb_class0, *d_posteriorProb_class1;

    double posteriorProb_class0[DocWords*DocNum];
    double posteriorProb_class1[DocWords*DocNum];

    // posteriorProb class 0 ---------------------

    int size_of_docClass = DocClass_0;

    hipMalloc((void **) &d_termInClass_0, DocNum*DocWords*sizeof(int));
    hipMalloc((void **) &d_nDoc_class, sizeof(int));
    hipMalloc((void **) &d_posteriorProb_class0, (DocWords*DocNum)*sizeof(double));

    hipMemcpy(d_termInClass_0, &termInClass_0, DocNum*DocWords*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_nDoc_class, &size_of_docClass, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_posteriorProb_class0, &posteriorProb_class0, (Classes*DocWords*DocNum)*sizeof(double), hipMemcpyHostToDevice);

    find_posterior<<<1,DocNum*DocWords>>>(d_termInClass_0, d_nDoc_class, d_posteriorProb_class0);

    hipMemcpy(&posteriorProb_class0, d_posteriorProb_class0, (DocWords*DocNum)*sizeof(double), hipMemcpyDeviceToHost);

    // hipFree(d_termInClass_0);
    hipFree(d_nDoc_class);
    // hipFree(d_posteriorProb_class0);
        
    // -------------------------------------------

    // cout << "----------" << endl;

    // class 1 -----------------------------------

    size_of_docClass = DocClass_1;

    hipMalloc((void **) &d_termInClass_1, DocNum*DocWords*sizeof(int));
    hipMalloc((void **) &d_nDoc_class, sizeof(int));
    hipMalloc((void **) &d_posteriorProb_class1, (DocWords*DocNum)*sizeof(double));

    hipMemcpy(d_termInClass_1, &termInClass_1, DocNum*DocWords*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_nDoc_class, &size_of_docClass, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_posteriorProb_class1, &posteriorProb_class1, (DocWords*DocNum)*sizeof(double), hipMemcpyHostToDevice);

    find_posterior<<<1,DocNum*DocWords>>>(d_termInClass_1, d_nDoc_class, d_posteriorProb_class1);

    hipMemcpy(&posteriorProb_class1, d_posteriorProb_class1, (DocWords*DocNum)*sizeof(double), hipMemcpyDeviceToHost);

    // hipFree(d_termInClass_1);
    hipFree(d_nDoc_class);
    // hipFree(d_posteriorProb_class1);

    // --------------------------------------------

    // show value of priorProb and posteriorProb

    // cout << endl <<"This is priorProb" << endl << endl;

    // for (int pp = 0 ; pp < Classes; pp++) {
    //     cout << priorProb[pp] << endl;
    // }

    // cout << endl << "this is posteriorProb" << endl << endl;

    // cout << "Class 0" << endl << endl;
    
    // for (int p = 0; p < DocWords*DocNum; p++) {
    //     cout << posteriorProb_class0[p] << endl;
    // }

    // cout << endl << "Class 1" << endl << endl;

    // for (int pd = 0; pd < DocWords*DocNum; pd++) {
    //     cout << posteriorProb_class1[pd] << endl;
    // }

    int docWord_arr[DocWords];

    //translate
    translateDocClassify(vocabList, doc, docWord_arr);
    //then get docWord_arr

    int *d_in;
    int *d_out;
    
	int h_out[1];

	hipMalloc((void**) &d_in, DocWords*sizeof(int));
	hipMalloc((void**) &d_out, sizeof(int));
    hipMemcpy(d_in, &docWord_arr, DocWords*sizeof(int), hipMemcpyHostToDevice);

    classifyperthread<<<1, DocWords*DocNum>>>(d_in, d_out, docWord_arr, vocabList.size(), d_posteriorProb_class0, d_posteriorProb_class1);

    hipMemcpy(&posteriorProb_class0, d_posteriorProb_class0, (DocWords*DocNum)*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&posteriorProb_class1, d_posteriorProb_class1, (DocWords*DocNum)*sizeof(double), hipMemcpyDeviceToHost);
    cout << "Class = " << findMax(priorProb, posteriorProb_class0, posteriorProb_class1, vocabList.size()) << endl;

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_termInClass_0);
    hipFree(d_termInClass_1);

    hipFree(d_termInClass_0);
    hipFree(d_posteriorProb_class0);

    hipFree(d_termInClass_1);
    hipFree(d_posteriorProb_class1);

	hipFree(d_in);
	hipFree(d_out);

	printf("time used: %f\n", milliseconds);
	
	return -1;
}